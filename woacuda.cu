#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <sstream>
#include <vector>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <limits>
#include <fstream>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <unistd.h>
#include <ctime>
#include <cstdlib>
#include <stdint.h>
#include <cstdio>
#include <sys/mman.h>
#include <hip/hip_cooperative_groups.h>
#define PI_F 3.141592654f


struct joints{
    float jointsval[6];
    bool ch = false;
    float ph = 0.0;
};

struct boundaries{
    float joint1b[2];
    float joint2b[2];
    float joint3b[2];
    float joint4b[2];
    float joint5b[2];
    float joint6b[2];
};


//////////DEVICE FUNCTIONS 

__device__ __forceinline__ float atomicMul(float* address, float val)
{
  int32_t* address_as_int = reinterpret_cast<int32_t*>(address);
  int32_t old = *address_as_int, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_int, assumed, __float_as_int(val * __int_as_float(assumed)));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ static float fatomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,__float_as_int(fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ static float fatomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,__float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ void fobj(joints currpos,float* tmpscore)
{
  *tmpscore=sqrt(currpos.jointsval[0]*currpos.jointsval[0]+currpos.jointsval[1]*currpos.jointsval[1]+currpos.jointsval[2]*currpos.jointsval[2]+currpos.jointsval[3]*currpos.jointsval[3]+currpos.jointsval[4]*currpos.jointsval[4]+currpos.jointsval[5]*currpos.jointsval[5]);
}

__global__ void WaoCycle(boundaries limit,int n_cycles,float* bestscore,joints* bestjoint,float factor)//estrai best ogni ciclo in host
{
  joints Leader_pos;
  float tmpscore;
  float a,a2,A,C,b,l,D_X_rand,X_rand;
  bool chkl1,chkl2,chkl3,chkl4,chkl5,chkl6,chku1,chku2,chku3,chku4,chku5,chku6;
  
  extern __shared__ joints shmem[];
  joints * jointshar  = (joints *)&shmem;
  
  hiprandState_t state;
  hiprand_init(clock64() ,threadIdx.x, 0, &state);
  
  jointshar[threadIdx.x].jointsval[0]=hiprand_uniform(&state)*(limit.joint1b[1]-limit.joint1b[0])+limit.joint1b[0];
  jointshar[threadIdx.x].jointsval[1]=hiprand_uniform(&state)*(limit.joint2b[1]-limit.joint2b[0])+limit.joint2b[0];
  jointshar[threadIdx.x].jointsval[2]=hiprand_uniform(&state)*(limit.joint3b[1]-limit.joint3b[0])+limit.joint3b[0];
  jointshar[threadIdx.x].jointsval[3]=hiprand_uniform(&state)*(limit.joint4b[1]-limit.joint4b[0])+limit.joint4b[0];
  jointshar[threadIdx.x].jointsval[4]=hiprand_uniform(&state)*(limit.joint5b[1]-limit.joint5b[0])+limit.joint5b[0];
  jointshar[threadIdx.x].jointsval[5]=hiprand_uniform(&state)*(limit.joint6b[1]-limit.joint6b[0])+limit.joint6b[0];
  for(int cyc=0;cyc<n_cycles;cyc++)
  {
    chkl1 = (jointshar[threadIdx.x].jointsval[0]<limit.joint1b[0]);
    chkl2 = (jointshar[threadIdx.x].jointsval[1]<limit.joint2b[0]);
    chkl3 = (jointshar[threadIdx.x].jointsval[2]<limit.joint3b[0]);
    chkl4 = (jointshar[threadIdx.x].jointsval[3]<limit.joint4b[0]);
    chkl5 = (jointshar[threadIdx.x].jointsval[4]<limit.joint5b[0]);
    chkl6 = (jointshar[threadIdx.x].jointsval[5]<limit.joint6b[0]);
    chku1 = (jointshar[threadIdx.x].jointsval[0]>limit.joint1b[1]);
    chku2 = (jointshar[threadIdx.x].jointsval[1]>limit.joint2b[1]);
    chku3 = (jointshar[threadIdx.x].jointsval[2]>limit.joint3b[1]);
    chku4 = (jointshar[threadIdx.x].jointsval[3]>limit.joint4b[1]);
    chku5 = (jointshar[threadIdx.x].jointsval[4]>limit.joint5b[1]);
    chku6 = (jointshar[threadIdx.x].jointsval[5]>limit.joint6b[1]);
    jointshar[threadIdx.x].jointsval[0] = jointshar[threadIdx.x].jointsval[0]*(!(chkl1+chku1))+(chkl1*limit.joint1b[0])+(chku1*limit.joint1b[1]);
    jointshar[threadIdx.x].jointsval[1] = jointshar[threadIdx.x].jointsval[1]*(!(chkl2+chku2))+(chkl2*limit.joint2b[0])+(chku2*limit.joint2b[1]);
    jointshar[threadIdx.x].jointsval[2] = jointshar[threadIdx.x].jointsval[2]*(!(chkl3+chku3))+(chkl3*limit.joint3b[0])+(chku3*limit.joint3b[1]);
    jointshar[threadIdx.x].jointsval[3] = jointshar[threadIdx.x].jointsval[3]*(!(chkl4+chku4))+(chkl4*limit.joint4b[0])+(chku4*limit.joint4b[1]);
    jointshar[threadIdx.x].jointsval[4] = jointshar[threadIdx.x].jointsval[4]*(!(chkl5+chku5))+(chkl5*limit.joint5b[0])+(chku5*limit.joint5b[1]);
    jointshar[threadIdx.x].jointsval[5] = jointshar[threadIdx.x].jointsval[5]*(!(chkl6+chku6))+(chkl6*limit.joint6b[0])+(chku6*limit.joint6b[1]);
    
    fobj(jointshar[threadIdx.x],&tmpscore);
//     printf("thr %d tmpscore %f \n",threadIdx.x,tmpscore);
    __syncthreads();

    fatomicMin(bestscore,tmpscore);
    __syncthreads();

    if(*bestscore==tmpscore){
      *bestjoint=jointshar[threadIdx.x];
    }

    a  =  2-cyc*((2) /n_cycles);
    a2 = -1+cyc*((-1)/n_cycles);
    
    A=2*a*hiprand_uniform(&state)-a;
    C=2*hiprand_uniform(&state);
  
    b=1;
    l=(a2-1)*hiprand_uniform(&state)+1;
    
    #pragma unroll
    for(int j=0;j<6;j++)
    {
      if(hiprand_uniform(&state)<0.5)
      {
        if(fabsf(A)>=factor)
        {
          X_rand = jointshar[static_cast<int>(floor((blockDim.x)*hiprand_uniform(&state)))].jointsval[j]; //blockDim.y e z??
          D_X_rand=abs(C*X_rand-jointshar[threadIdx.x].jointsval[j]);
          jointshar[threadIdx.x].jointsval[j]=X_rand-A*D_X_rand;
        }
        else
        {
          jointshar[threadIdx.x].jointsval[j] = Leader_pos.jointsval[j]-A*abs(C*Leader_pos.jointsval[j]-jointshar[threadIdx.x].jointsval[j]);
        }
      }
      else
      {  
        jointshar[threadIdx.x].jointsval[j] = abs(Leader_pos.jointsval[j]-jointshar[threadIdx.x].jointsval[j])*exp(b*l)*cos(l*2*PI_F)+Leader_pos.jointsval[j];  
      }
    }
    if(threadIdx.x==0) printf("best score : %f \n",*bestscore);
  }
}


///////////CLASS


class WoaCuda
{
  int n_whales;
  int n_cycles;
  int n_joints;
  float factor;
  
  float *deviceBestscore;
  
  joints *hostjointbest = static_cast<joints*>(malloc(sizeof(joints)));
  joints *devicejointbest;

  boundaries jointlimits;
  
  size_t shrbytes;
  
  public:

    float *hostBestscore  = static_cast<float *>(malloc(sizeof(float)));
    
    WoaCuda(int nwhales,int ncyc,boundaries limits,float factor);
    void RunCycle();
    void Copytohost();
};

///////////CLASS METHODS
WoaCuda::WoaCuda(int nwhales,int ncyc,boundaries limits,float factor)
{
  n_whales=nwhales;
  n_cycles=ncyc;
  jointlimits=limits;
  
  shrbytes=n_whales*sizeof(joints);
  
  hipMalloc(static_cast<float**>(&deviceBestscore),sizeof(float));
  memset(hostBestscore,0,sizeof(float));
  *hostBestscore=std::numeric_limits<float>::infinity();
  hipMemcpy(deviceBestscore,hostBestscore,sizeof(float),hipMemcpyHostToDevice);
  hipMalloc(static_cast<joints**>(&devicejointbest),sizeof(joints));
  memset(hostjointbest,0,sizeof(joints));
  hipMemcpy(devicejointbest,hostjointbest,sizeof(joints),hipMemcpyHostToDevice);
  
  printf("whale number: %lu \n",n_whales);
  printf("cycles      : %lu \n",n_cycles);
  printf("shared bytes: %lu \n",shrbytes);
}

void WoaCuda::RunCycle() //launch cuda kernel 
{
  WaoCycle<<<1,n_whales,shrbytes>>>(jointlimits,n_cycles,deviceBestscore,devicejointbest,factor);//<<<blocks,thread>>>
  if (hipSuccess != hipDeviceSynchronize()) {
    printf("ERROR in WaoCycle\n");
    exit(-2);
  }
}

void WoaCuda::Copytohost()
{
  hipMemcpy(hostBestscore,deviceBestscore,sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(hostjointbest,devicejointbest,sizeof(joints),hipMemcpyDeviceToHost);
}


///////////MAIN

int main(int argc, char *argv[]){
  boundaries limit;
  limit.joint1b[0]=-30;
  limit.joint1b[1]= 30;
  limit.joint2b[0]=-40;
  limit.joint2b[1]= 40;
  limit.joint3b[0]=-40;
  limit.joint3b[1]= 40;
  limit.joint4b[0]=-360;
  limit.joint4b[1]= 360;
  limit.joint5b[0]=-360;
  limit.joint5b[1]= 360;
  limit.joint6b[0]=-360;
  limit.joint6b[1]= 360;
  
  int whl=atof(argv[1]);
  int ncyc=atof(argv[2]);
  int fact=1;
  
  WoaCuda testwao(whl,ncyc,limit,fact);//nwhales,cycles,limits,joints
  testwao.RunCycle(); 
  testwao.Copytohost();
   printf("best: %f\n\n",*(testwao.hostBestscore));
  return 0;
}


